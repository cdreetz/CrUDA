#include "hip/hip_runtime.h"
// src/kernels/matmul_naive.cu
#include "kernels/matmul_naive.cuh"

namespace matmul {

__global__ void naiveMatrixMultiply(
  const float* A,
  const float* B,
  float* C,
  int M, int N, int K
) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float sum = 0.0f;
    for (int k = 0; k < K; k++) {
      sum += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

} // namespace matmul
